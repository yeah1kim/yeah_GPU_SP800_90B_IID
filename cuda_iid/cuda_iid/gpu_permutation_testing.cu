
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <hiprand/hiprand_kernel.h>
#include "header.h"
#include "kernel_functions.cuh"

int gpu_permutation_testing(double *dGPU_runtime, uint32_t *counts, double *results, double mean, double median,
	uint8_t *data, uint32_t size, uint32_t len, uint32_t N, uint32_t num_block, uint32_t num_thread)
{
	uint32_t loop = 10000 / N;
	if ((10000 % N) != 0)	loop++;
	hipError_t cudaStatus;
	uint8_t *dev_data, *dev_Ndata;
	double *dev_results;
	hiprandState *dev_curand;
	uint32_t *dev_cnt;
	hipEvent_t cuda_time_start, cuda_time_end;
	hipEventCreate(&cuda_time_start);
	hipEventCreate(&cuda_time_end);

	uint32_t blen;
	uint8_t *dev_bNdata;
	if (size == 1) {
		blen = len / 8;
		if ((len % 8) != 0)	blen++;
	}

	/* choose which GPU to run on. */
	CUDA_ERRORCHK((hipSetDevice(0) != hipSuccess));

	/* allocate memory on the GPU. */
	size_t Nlen = N * len;
	size_t Nblen = N * blen;
	CUDA_CALLOC_ERRORCHK((hipMalloc((void**)&dev_data, len * sizeof(uint8_t)) != hipSuccess));
	CUDA_CALLOC_ERRORCHK((hipMalloc((void**)&dev_Ndata, Nlen * sizeof(uint8_t)) != hipSuccess));
	CUDA_CALLOC_ERRORCHK((hipMalloc((void**)&dev_results, 18 * sizeof(double)) != hipSuccess));
	CUDA_CALLOC_ERRORCHK((hipMalloc((void**)&dev_curand, N * sizeof(hiprandState)) != hipSuccess));
	CUDA_CALLOC_ERRORCHK((hipMalloc((void**)&dev_cnt, 54 * sizeof(uint32_t)) != hipSuccess));
	if (size == 1)
		CUDA_CALLOC_ERRORCHK((hipMalloc((void**)&dev_bNdata, Nblen * sizeof(uint8_t)) != hipSuccess));

	/* copy data from the CPU to the GPU. */
	CUDA_MEMCPY_ERRORCHK((hipMemcpy(dev_data, data, len * sizeof(uint8_t), hipMemcpyHostToDevice) != hipSuccess));
	CUDA_MEMCPY_ERRORCHK((hipMemcpy(dev_results, results, 18 * sizeof(double), hipMemcpyHostToDevice) != hipSuccess));
	CUDA_MEMCPY_ERRORCHK((hipMemcpy(dev_cnt, counts, 54 * sizeof(uint32_t), hipMemcpyHostToDevice) != hipSuccess));

	/* start the CUDA timer. */
	hipEventRecord(cuda_time_start, 0);

	/* initialize the seeds used by hiprand() function. */
	setup_curand_kernel << < num_block, num_thread >> > (dev_curand, (uint32_t)time(NULL));
	hipDeviceSynchronize();

	for (int i = 0; i < loop; i++) {
		if (size == 1) {
			binary_shuffling_kernel << < num_block, num_thread >> > (dev_Ndata, dev_bNdata, dev_data, dev_curand, len, blen, N);
			hipDeviceSynchronize();
			binary_b4_statistical_tests_kernel << < num_block * 4, num_thread >> > (dev_cnt, dev_results, mean, median, dev_Ndata, dev_bNdata, size, len, blen, N, num_block);
			hipDeviceSynchronize();
		}
		else {
			shuffling_kernel << < num_block, num_thread >> > (dev_Ndata, dev_data, dev_curand, len, N);
			hipDeviceSynchronize();
			b2_statistical_tests_kernel << < num_block * 2, num_thread >> > (dev_cnt, dev_results, mean, median, dev_Ndata, size, len, N, num_block);
			hipDeviceSynchronize();
		}

		/* copy data from the GPU to the CPU. */
		CUDA_MEMCPY_ERRORCHK((hipMemcpy(counts, dev_cnt, 54 * sizeof(uint32_t), hipMemcpyDeviceToHost) != hipSuccess));
		uint8_t check = 0;
		for (int t = 0; t < 18; t++) {
			if (((counts[3 * t] + counts[3 * t + 1]) > 5) && ((counts[3 * t + 1] + counts[3 * t + 2]) > 5))
				check++;
		}
		if (check == 18)
			break;
	}

	/* stop the CUDA timer. */
	hipEventRecord(cuda_time_end, 0);
	hipDeviceSynchronize();


	/* check for any errors launching the kernel. */
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "gpu_permutation_testing launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	/* hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch. */
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching gpu_permutation_testing!\n", cudaStatus);
		goto Error;
	}

	/* calculate the run-time of the permutation testing (measured by CUDA timer) */
	float cuda_time = 0;
	hipEventElapsedTime(&cuda_time, cuda_time_start, cuda_time_end);
	*dGPU_runtime = (double)cuda_time;


	/* cuda device reset. */
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	
Error:
	hipFree(dev_data);
	hipFree(dev_Ndata);
	hipFree(dev_results);
	hipFree(dev_curand);
	hipFree(dev_cnt);
	if (size == 1) 
		hipFree(dev_bNdata);
	hipEventDestroy(cuda_time_start);
	hipEventDestroy(cuda_time_end);
	
	return 0;
}