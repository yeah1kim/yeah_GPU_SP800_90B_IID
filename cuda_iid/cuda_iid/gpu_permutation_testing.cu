#include "hip/hip_runtime.h"
/*
 * GPU-based parallel implementation of the IID test of NIST SP 800-90B.
 *
 * Copyright(C) < 2020 > <Yewon Kim>
 *
 * This program is free software : you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with this program.If not, see < https://www.gnu.org/licenses/>.
 */


#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <hiprand/hiprand_kernel.h>
#include "header.h"
#include "kernel_functions.cuh"

 /**
  * @brief Perform 10,000 iterations in parallel on the GPU. (exclude the compression test)
  *  - That is, perform {$N} iterations in the GPU and repeat ceil(10,000 / $N) times.
  *  - In each iteration, the original data are shuffled, 18 statistical tests are performed on the shuffled data,
  *    and the results are compared with the original test statistics.
  * @param double $dGPU_runtime: Runtime of 10,000 iterations measured by CUDA timer
  * @param uint32_t $counts[]: The counters, that is original test statistics's rankings
  * @param double $results[]: The results of 19 statistical tests on the original data
  * @param double $mean: Mean value of the original data(input)
  * @param double $median: Median value of the original data(input)
  * @param uint8_t $data[]: The original data(input), which consists of (noise) samples
  * @param uint32_t $size: The size of sample in bits (1~8)
  * @param uint32_t $len: The number of samples in the original data
  * @param uint32_t $N: The number of iterations processing in parallel on the GPU
  * @param uint32_t $num_block: The number of CUDA blocks
  * @param uint32_t $num_thread: The number of CUDA threads per block
  * @return bool $iid_check_result
  */
bool gpu_permutation_testing(double *dgpu_runtime, uint32_t *counts, double *results, double mean, double median,
	uint8_t *data, uint32_t size, uint32_t len, uint32_t N, uint32_t num_block, uint32_t num_thread)
{
	int32_t i;
	uint8_t num_runtest = 0;
	uint32_t loop = 10000 / N;
	if ((10000 % N) != 0)	loop++;
	uint32_t blen;
	if (size == 1) {
		blen = len / 8;
		if ((len % 8) != 0)	blen++;
	}
	size_t Nlen = N * len;
	size_t Nblen = N * blen;

	hipError_t cudaStatus;
	uint8_t *dev_data;
	uint8_t *dev_Ndata, *dev_bNdata;
	double *dev_results;
	hiprandState *dev_curand;
	uint32_t *dev_cnt;
	hipEvent_t cuda_time_start, cuda_time_end;
	hipEventCreate(&cuda_time_start);
	hipEventCreate(&cuda_time_end);

	/* choose which GPU to run on. */
	CUDA_ERRORCHK((hipSetDevice(0) != hipSuccess));

	/* allocate memory on the GPU. */
	CUDA_CALLOC_ERRORCHK((hipMalloc((void**)&dev_data, len * sizeof(uint8_t)) != hipSuccess));
	CUDA_CALLOC_ERRORCHK((hipMalloc((void**)&dev_Ndata, Nlen) != hipSuccess));
	CUDA_CALLOC_ERRORCHK((hipMalloc((void**)&dev_results, 18 * sizeof(double)) != hipSuccess));
	CUDA_CALLOC_ERRORCHK((hipMalloc((void**)&dev_curand, N * sizeof(hiprandState)) != hipSuccess));
	CUDA_CALLOC_ERRORCHK((hipMalloc((void**)&dev_cnt, 54 * sizeof(uint32_t)) != hipSuccess));
	if (size == 1)
		CUDA_CALLOC_ERRORCHK((hipMalloc((void**)&dev_bNdata, Nblen * sizeof(uint8_t)) != hipSuccess));

	/* copy data from the CPU to the GPU. */
	CUDA_MEMCPY_ERRORCHK((hipMemcpy(dev_data, data, len * sizeof(uint8_t), hipMemcpyHostToDevice) != hipSuccess));
	CUDA_MEMCPY_ERRORCHK((hipMemcpy(dev_results, results, 18 * sizeof(double), hipMemcpyHostToDevice) != hipSuccess));
	CUDA_MEMCPY_ERRORCHK((hipMemcpy(dev_cnt, counts, 54 * sizeof(uint32_t), hipMemcpyHostToDevice) != hipSuccess));

	/* start the CUDA timer. */
	hipEventRecord(cuda_time_start, 0);

	/* initialize the seeds used by hiprand() function. */
	setup_curand_kernel << < num_block, num_thread >> > (dev_curand, (uint32_t)time(NULL));

	/* generate {$N} shuffled data by permuting the original data {$N} times in parallel.
	 * perform 18 statistical tests on each of {$N} shuffled data and compares the shuffled and original test statistics in parallel.
	 */
	for (i = 0; i < loop; i++) {
		if (size == 1) {
			binary_shuffling_kernel << < num_block, num_thread >> > (dev_Ndata, dev_bNdata, dev_data, dev_curand, len, blen, N);
			binary_statistical_tests_kernel << < num_block * 4, num_thread >> > (dev_cnt, dev_results, mean, median, dev_Ndata, dev_bNdata, size, len, blen, N, num_block);

			/* copy data from the GPU to the CPU. */
			CUDA_MEMCPY_ERRORCHK((hipMemcpy(counts, dev_cnt, 54 * sizeof(uint32_t), hipMemcpyDeviceToHost) != hipSuccess));
			num_runtest = 0;
			for (int t = 0; t < 18; t++) {
				if (((counts[3 * t] + counts[3 * t + 1]) > 5) && ((counts[3 * t + 1] + counts[3 * t + 2]) > 5))
					num_runtest++;
			}
			if (num_runtest == 18)
				break;
		}
		else {
			shuffling_kernel << < num_block, num_thread >> > (dev_Ndata, dev_data, dev_curand, len, N);
			statistical_tests_kernel << < num_block * 2, num_thread >> > (dev_cnt, dev_results, mean, median, dev_Ndata, size, len, N, num_block);

			/* copy data from the GPU to the CPU. */
			CUDA_MEMCPY_ERRORCHK((hipMemcpy(counts, dev_cnt, 54 * sizeof(uint32_t), hipMemcpyDeviceToHost) != hipSuccess));
			num_runtest = 0;
			for (int t = 0; t < 18; t++) {
				if (((counts[3 * t] + counts[3 * t + 1]) > 5) && ((counts[3 * t + 1] + counts[3 * t + 2]) > 5))
					num_runtest++;
			}
			if (num_runtest == 18)
				break;
		}
	}

	/* stop the CUDA timer. */
	hipEventRecord(cuda_time_end, 0);
	hipDeviceSynchronize();

	/* check for any errors launching the kernel. */
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "gpu_permutation_testing launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	/* hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch. */
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching gpu_permutation_testing!\n", cudaStatus);
		goto Error;
	}

	/* calculate the run-time of the permutation testing (measured by CUDA timer) */
	float cuda_time = 0;
	hipEventElapsedTime(&cuda_time, cuda_time_start, cuda_time_end);
	*dgpu_runtime = (double)cuda_time;

Error:
	hipFree(dev_data);
	hipFree(dev_Ndata);
	hipFree(dev_results);
	hipFree(dev_curand);
	hipFree(dev_cnt);
	if (size == 1)
		hipFree(dev_bNdata);
	hipEventDestroy(cuda_time_start);
	hipEventDestroy(cuda_time_end);

	if (num_runtest == 18) // IID
		return true;
	else // Non-IID
		return false;
}